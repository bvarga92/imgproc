#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define NUM_ELEMENTS 100 //vektorok elemszama

/* CUDA kernel */
__global__ void vecadd(const int* a, const int* b, int* c){
	int thId=threadIdx.x;
	c[thId]=a[thId]+b[thId];
}

/* foprogram */
int main(){
	int a[NUM_ELEMENTS], b[NUM_ELEMENTS], c[NUM_ELEMENTS];
	int *bufferA, *bufferB, *bufferC;
	int i;
	/* ha tobb GPU van, akkor az elsot valasztjuk */
	hipSetDevice(0);
	/* kernel parameterek atadasa */
	srand(123);
	for(i=0;i<NUM_ELEMENTS;i++){
		a[i]=rand()%1000;
		b[i]=rand()%1000;
	}
	hipMalloc((void**)&bufferA,sizeof(int)*NUM_ELEMENTS);
	hipMalloc((void**)&bufferB,sizeof(int)*NUM_ELEMENTS);
	hipMalloc((void**)&bufferC,sizeof(int)*NUM_ELEMENTS);
	hipMemcpy(bufferA,a,sizeof(int)*NUM_ELEMENTS,hipMemcpyHostToDevice);
	hipMemcpy(bufferB,b,sizeof(int)*NUM_ELEMENTS,hipMemcpyHostToDevice);
	/* kernel futtatasa NUM_ELEMENTS darab szallal */
	vecadd<<<1,NUM_ELEMENTS>>>(bufferA,bufferB,bufferC);
	if (hipGetLastError() != hipSuccess) {
		printf("Error: failed to launch kernel.\n");
		hipFree(bufferA);
		hipFree(bufferB);
		hipFree(bufferC);
		return 1;
	}
	hipDeviceSynchronize();
	/* eredmeny kiolvasasa */
	hipMemcpy(c,bufferC,sizeof(int)*NUM_ELEMENTS,hipMemcpyDeviceToHost);
	/* befejezes */
	hipFree(bufferA);
	hipFree(bufferB);
	hipFree(bufferC);
	printf("\n");
	for(i=0;i<NUM_ELEMENTS;i++) printf("%3d + %3d = %4d\n",a[i],b[i],c[i]);
	getchar();
	hipDeviceReset();
	return 0;
}
